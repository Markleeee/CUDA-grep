#include "hip/hip_runtime.h"

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "pnfa.h"

#define PRINT(time,...) if(!time) printf(__VA_ARGS__)

#define IS_EMPTY(l) (l->n == 0)
#define PUSH(l, state) l->s[l->n++] = state
#define POP(l) l->s[l->n]; 



__device__ inline void paddstate(List*, State*, List*, int *);
__device__ inline void pstep(List*, int, List*, int *);

/* Compute initial state list */
__device__ inline List*
pstartlist(State *start, List *l, int *dlistid)
{
	l->n = 0;
	(*dlistid)++;

	List addStartState;
	paddstate(l, start, &addStartState, dlistid);
	return l;
}

/* Check whether state list contains a match. */
__device__ inline int
ispmatch(List *l)
{
	int i;

	for(i=0; i<l->n; i++) {
		if(l->s[i]->c == 256)
			return 1;
	}
	return 0;
}

/* Add s to l, following unlabeled arrows. */
	__device__ inline void
paddstate(List *l, State *s, List *addStateList, int *dlistid)
{	
	addStateList->n = 0;
	PUSH(addStateList, s);
	/* follow unlabeled arrows */
	while(!IS_EMPTY(addStateList)) {	
	
		addStateList->n--;
		s = POP(addStateList);
	
		// lastlist check is present to ensure that if
		// multiple states point to this state, then only
		//one instance of the state is added to the list
		if(s == NULL);
		else if (s->c == 257) {
			s->lastlist = *dlistid; 
			PUSH(addStateList, s->out);
			PUSH(addStateList, s->out1);	
		}
		else {
			s->lastlist = *dlistid; 
			l->s[l->n++] = s;
		}
	}
}

/*
 * pstep the NFA from the states in clist
 * past the character c,
 * to create next NFA state set nlist.
 */
__device__ inline void
pstep(List *clist, int c, List *nlist, int *dlistid)
{
	int i;
	State *s;
	(*dlistid)++;
	nlist->n = 0;
	for(i=0; i<clist->n; i++){
		s = clist->s[i];
	
		if(s->c == c || s->c == Any){
			List addStartState;
			paddstate(nlist, s->out, &addStartState, dlistid);
		}
	}
}

/* Run NFA to determine whether it matches s. */
__device__ inline int
pmatch(State *start, char *s, List *dl1, List *dl2, int * dlistid)
{
	int c;
	List *clist, *nlist, *t;

	clist = pstartlist(start, dl1, dlistid);
	nlist = dl2;
	for(; *s; s++){
		c = *s & 0xFF;
		pstep(clist, c, nlist, dlistid);
		t = clist; clist = nlist; nlist = t;	// swap clist, nlist 
	
		// check for a match in the middle of the string
		if (ispmatch(clist))
			return 1;

	}
	return ispmatch(clist);
}

/* Check for a string match at all possible start positions */
__device__ inline int panypmatch(State *start, char *s, List *dl1, List *dl2, int *dlistid) { 
	int isMatch = pmatch(start, s, dl1, dl2, dlistid);
	int index = 0;
	int len = 0; 
	char * sc = s;
	while(*sc != 0) {
		len ++;
		sc += 1;	
	}
	
	while (!isMatch && index < len) {
		isMatch = pmatch(start, s + index, dl1, dl2, dlistid);
		index ++;
	}
	return isMatch;
}


/* Allocate and initialize State */
__device__ inline State* pstate(int c, State *out, State *out1)
{
	State *s = (states + pnstate); // assign a state
	
	s->id = pnstate++;
	s->lastlist = 0;
	s->c = c;
	s->out = out;
	s->out1 = out1;
	
	// device pointer of itself
	// serves no real purpose other than to help transfer the NFA over
	s->dev = NULL;
	
	s->free = 0;
	return s;
}


/* Initialize frag struct. */
__device__ inline Frag
pfrag(State *start, Ptrlist *out)
{
	Frag n = { start, out };
	return n;
}

/* Create singleton list containing just outp. */
__device__ inline Ptrlist*
plist1(State **outp)
{
	Ptrlist *l;

	l = (Ptrlist*)outp;
	l->next = NULL;
	return l;
}

/* Patch the list of states at out to point to start. */
__device__ inline void
ppatch(Ptrlist *l, State *s)
{
	Ptrlist *next;

	for(; l; l=next){
		next = l->next;
		l->s = s;
	}
}

/* Join the two lists l1 and l2, returning the combination. */
__device__ inline Ptrlist*
pappend(Ptrlist *l1, Ptrlist *l2)
{
	Ptrlist *oldl1;

	oldl1 = l1;
	while(l1->next)
		l1 = l1->next;
	l1->next = l2;
	return oldl1;
}


/*
 * Convert postfix regular expression to NFA.
 * Return start state.
 */
 
__device__ inline State*
ppost2nfa(char *postfix)
{
	char *p;
	Frag stack[1000], *stackp, e1, e2, e;
	State *s;

	// fprintf(stderr, "postfix: %s\n", postfix);

	if(postfix == NULL)
		return NULL;

#define push(s) *stackp++ = s
#define pop() *--stackp

	stackp = stack;
	for(p=postfix; *p; p++){
		switch(*p){
            case ANY: /* any (.) */
				s = pstate(Any, NULL, NULL);
				push(pfrag(s, plist1(&s->out)));
				break;
			default:
				s = pstate(*p, NULL, NULL);
				push(pfrag(s, plist1(&s->out)));
				break;
			case CONCATENATE:	/* catenate */
				e2 = pop();
				e1 = pop();
				ppatch(e1.out, e2.start);
				push(pfrag(e1.start, e2.out));
				break;
			case ALTERNATE:	/* alternate (|)*/
				e2 = pop();
				e1 = pop();
				s = pstate(Split, e1.start, e2.start);
				push(pfrag(s, pappend(e1.out, e2.out)));
				break;
			case QUESTION:	/* zero or one (?)*/
				e = pop();
				s = pstate(Split, e.start, NULL);
				push(pfrag(s, pappend(e.out, plist1(&s->out1))));
				break;
			case STAR:	/* zero or more (*)*/
				e = pop();
				s = pstate(Split, e.start, NULL);
				ppatch(e.out, s);
				push(pfrag(s, plist1(&s->out1)));
				break;
			case PLUS:	/* one or more (+)*/
				e = pop();
				s = pstate(Split, e.start, NULL);
				ppatch(e.out, s);
				push(pfrag(e.start, plist1(&s->out1)));
				break;
		}
	}

	e = pop();
	if(stackp != stack)
		return NULL;

	ppatch(e.out, &pmatchstate);

	return e.start;
#undef pop
#undef push
}






__global__ void parallelMatch(char * bigLine, u32 * tableOfLineStarts, int lineIndex, int nstate, int time, char *postfix, unsigned char * devResult) {

	State s[100];
	pnstate = 0;
	states = s;

	State *st = ppost2nfa(postfix);

	List d1;
	List d2;	
	int dlistid;


	int i;
	for (i = blockIdx.x * blockDim.x + threadIdx.x; i < lineIndex; i += gridDim.x * blockDim.x) { 
       
        char * lineSegment = bigLine + tableOfLineStarts[i];

        if (panypmatch(st, lineSegment, &d1, &d2, &dlistid)) 
			devResult[i] = 1;
		else
			devResult[i] = 0;
	}
}

void pMatch(char * bigLine, u32 * tableOfLineStarts, int lineIndex, int nstate, int time, char * postfix, char **lines) {

	unsigned char *devResult;
	hipMalloc(&devResult, lineIndex * sizeof(unsigned char));
	
	parallelMatch<<<256, 256>>>(bigLine, tableOfLineStarts, lineIndex, nstate ,time, postfix, devResult);
	
	hipDeviceSynchronize();

	//TODO free states
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(error));
        exit(-1);
    }

	unsigned char *hostResult = (unsigned char *) malloc (lineIndex * sizeof(unsigned char));
	hipMemcpy(hostResult, devResult, lineIndex * sizeof(unsigned char), hipMemcpyDeviceToHost);
	for (int i = 0; i < lineIndex; i++) {
		if(hostResult[i] == 1) 
			printf("%s", lines[i]);
	}

	hipFree(&bigLine);
    hipFree(&tableOfLineStarts);

}


// taken from 15-418 assignment 2
void
printCudaInfo() {
    
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    
    printf("Found %d CUDA devices\n", deviceCount);
    
    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }

}
