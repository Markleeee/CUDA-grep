#include "hip/hip_runtime.h"

#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "pnfa.h"

#define PRINT(time,...) if(!time) printf(__VA_ARGS__)

#define IS_EMPTY(l) (l->n == 0)
#define PUSH(l, state) l->s[l->n++] = state
#define POP(l) l->s[l->n]; 


__device__ static int dlistid;
__device__ State pmatchstate = { Match };	/* matching state */


__device__ inline void paddstate(List*, State*, List*);
__device__ inline void pstep(List*, int, List*);

/* Compute initial state list */
__device__ inline List*
pstartlist(State *start, List *l)
{
	l->n = 0;
	dlistid++;

	List addStartState;
	paddstate(l, start, &addStartState);
	return l;
}

/* Check whether state list contains a match. */
__device__ inline int
ispmatch(List *l)
{
	int i;

	for(i=0; i<l->n; i++) {
		if(l->s[i]->c == 256)
			return 1;
	}
	return 0;
}

/* Add s to l, following unlabeled arrows. */
	__device__ inline void
paddstate(List *l, State *s, List *addStateList)
{	
	addStateList->n = 0;
	PUSH(addStateList, s);
	/* follow unlabeled arrows */
	while(!IS_EMPTY(addStateList)) {	
	
		addStateList->n--;
		s = POP(addStateList);
	
		// lastlist check is present to ensure that if
		// multiple states point to this state, then only
		//one instance of the state is added to the list
		if(s == NULL || s->lastlist == dlistid);
		else if (s->c == 257) {
			s->lastlist = dlistid; 
			PUSH(addStateList, s->out);
			PUSH(addStateList, s->out1);	
		}
		else {
			s->lastlist = dlistid; 
			l->s[l->n++] = s;
		}
	}
}

/*
 * pstep the NFA from the states in clist
 * past the character c,
 * to create next NFA state set nlist.
 */
__device__ inline void
pstep(List *clist, int c, List *nlist)
{
	int i;
	State *s;
	dlistid++;
	nlist->n = 0;
	for(i=0; i<clist->n; i++){
		s = clist->s[i];
	
		if(s->c == c || s->c == Any){
			List addStartState;
			paddstate(nlist, s->out, &addStartState);
		}
	}
}

/* Run NFA to determine whether it matches s. */
__device__ inline int
pmatch(State *start, char *s, List *dl1, List *dl2)
{
	int c;
	List *clist, *nlist, *t;

	clist = pstartlist(start, dl1);
	nlist = dl2;
	for(; *s; s++){
		c = *s & 0xFF;
		pstep(clist, c, nlist);
		t = clist; clist = nlist; nlist = t;	// swap clist, nlist 
	
		// check for a match in the middle of the string
		if (ispmatch(clist))
			return 1;

	}
	return ispmatch(clist);
}

/* Check for a string match at all possible start positions */
__device__ inline int panypmatch(State *start, char *s, List *dl1, List *dl2) { 
	int isMatch = pmatch(start, s, dl1, dl2);
	int index = 0;
	int len = 0; 
	char * sc = s;
	while(*sc != 0) {
		len ++;
		sc += 1;	
	}
	
	while (!isMatch && index < len) {
		isMatch = pmatch(start, s + index, dl1, dl2);
		index ++;
	}
	return isMatch;
}


__global__ void parallelMatch(State *start, char **lines, int lineIndex, int nstate, int time) {
	List d1;
	List d2;	


	int i;
	for (i = blockIdx.x * blockDim.x + threadIdx.x; i < lineIndex; i += gridDim.x * blockDim.x) { 
		if (panypmatch(start, lines[i], &d1, &d2)) 
			PRINT(time, "%s", lines[i]);
	}

/*
	// test to ensure that strings are copied over correctly
	for( int i = 0; i < lineIndex; i++) {
		printf("%s", lines[i]);		
	}
*/
	
}

void pMatch(State *start, char **lines, int lineIndex, int nstate, int time) {
		//printCudaInfo(); 
	parallelMatch<<<1,1>>>(start,lines,lineIndex, nstate ,time);


	//TODO free states

	int i;	
	for (i = 0; i <= lineIndex; i++) 
		hipFree(&(lines[i]));
	hipFree(&lines);

}


// taken from 15-418 assignment 2
void
printCudaInfo() {
    
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);
    
    printf("Found %d CUDA devices\n", deviceCount);
    
    for (int i=0; i<deviceCount; i++) {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }

}
